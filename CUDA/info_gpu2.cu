
#include "hip/hip_runtime.h"


#include <iostream>
using namespace std;

int main()
{
	int count;
	hipGetDeviceCount(&count);

	hipDeviceProp_t prop;

	for (int i = 0; i < count; ++i)
	{
		hipGetDeviceProperties(&prop, i);

		cout << "Device " << i << ": " << prop.name << endl;
		cout << "Compute Capability: " << prop.major << "." << prop.minor << endl;

	}

	return 0;
}


